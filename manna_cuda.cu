#include "hip/hip_runtime.h"
#include "params.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>

#if N < 65536u
typedef uint16_t MannaSizeType;
#else
typedef uint32_t MannaSizeType;
#endif

typedef uint32_t MannaItemType;

__device__ inline uint32_t xorshift32(uint32_t &state) {
    state ^= state << 13;
    state ^= state >> 17;
    state ^= state << 5;
    return state;
}

// GPU-based reduction to check if any block is active
__global__ void check_activity_kernel(uint32_t *active_mask, int num_blocks, int *global_active) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for block-level reduction
    __shared__ int block_has_active[256]; // Assuming max 256 threads per block
    
    // Each thread checks its assigned blocks
    bool thread_active = false;
    for (int i = idx; i < num_blocks; i += blockDim.x * gridDim.x) {
        if (active_mask[i] != 0) {
            thread_active = true;
            break;
        }
    }
    
    block_has_active[threadIdx.x] = thread_active ? 1 : 0;
    __syncthreads();
    
    // Block-level reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            block_has_active[threadIdx.x] = block_has_active[threadIdx.x] || block_has_active[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // First thread of each block writes to global memory
    if (threadIdx.x == 0 && block_has_active[0]) {
        atomicOr(global_active, 1);
    }
}

// Optimized destabilization kernel with better memory access patterns
__global__ void destab_kernel_opt(
    MannaItemType *h,
    MannaItemType *temp_h,
    uint32_t      *rng)
{
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= N) return;

    MannaItemType c = h[gid];
    
    // Early exit for inactive cells
    if (c <= 1) return;
    
    // Handle overflow case
    if (c >= 32) {
        h[gid] = 0;
        return;
    }

    h[gid] = 0;
    uint32_t r = xorshift32(rng[gid]);
    uint32_t mask = ((1u << c) - 1u) & r;
    uint32_t to_right = __popc(mask);
    uint32_t to_left  = c - to_right;

    unsigned right = (gid + 1) & (N - 1);
    unsigned left  = (gid + N - 1) & (N - 1);

    // Use non-atomic writes when possible (if we can guarantee no conflicts)
    atomicAdd(&temp_h[right], to_right);
    atomicAdd(&temp_h[left],  to_left);
}

// Optimized merge kernel with early termination
__global__ void merge_kernel_opt(
    MannaItemType *h,
    MannaItemType *temp_h,
    uint32_t      *active_mask)
{
    unsigned int gid  = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int lane = threadIdx.x & 31;

    MannaItemType new_val = 0;
    bool my_active = false;
    
    // Merge step
    if (gid < N) {
        new_val = h[gid] + temp_h[gid];
        h[gid] = new_val;
        my_active = (new_val > 1);
        temp_h[gid] = 0; // Clear for next iteration
    }

    // Warp-wide ballot for activity detection
    uint32_t warp_mask = __ballot_sync(0xFFFFFFFFu, my_active);

    // Use shared memory to avoid redundant atomic operations
    __shared__ bool block_active;
    if (threadIdx.x == 0) {
        block_active = false;
    }
    __syncthreads();
    
    if (lane == 0 && warp_mask != 0) {
        block_active = true;
    }
    __syncthreads();

    // Write block activity once per block
    if (threadIdx.x == 0) {
        active_mask[blockIdx.x] = block_active ? 1u : 0u;
    }
}

// Batch multiple iterations in a single kernel launch
__global__ void multi_step_kernel(
    MannaItemType *h,
    MannaItemType *temp_h,
    uint32_t      *rng_state,
    uint32_t      *active_mask,
    int           max_steps,
    int           *steps_taken)
{
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    for (int step = 0; step < max_steps; step++) {
        __syncthreads(); // Synchronize across all threads in grid (requires cooperative groups for multi-block)
        
        // Check if we should continue (simplified version)
        if (gid == 0 && step > 0) {
            // Simple activity check - in practice you'd need a more sophisticated approach
            bool any_active = false;
            for (int i = 0; i < gridDim.x && !any_active; i++) {
                if (active_mask[i]) any_active = true;
            }
            if (!any_active) {
                *steps_taken = step;
                return;
            }
        }
        
        // Reset temp array
        if (gid < N) {
            temp_h[gid] = 0;
        }
        if (gid < gridDim.x) {
            active_mask[gid] = 0;
        }
        __syncthreads();
        
        // Destabilization phase
        if (gid < N) {
            MannaItemType c = h[gid];
            if (c > 1 && c < 32) {
                h[gid] = 0;
                uint32_t r = xorshift32(rng_state[gid]);
                uint32_t mask = ((1u << c) - 1u) & r;
                uint32_t to_right = __popc(mask);
                uint32_t to_left = c - to_right;
                
                unsigned right = (gid + 1) & (N - 1);
                unsigned left = (gid + N - 1) & (N - 1);
                
                atomicAdd(&temp_h[right], to_right);
                atomicAdd(&temp_h[left], to_left);
            } else if (c >= 32) {
                h[gid] = 0;
            }
        }
        __syncthreads();
        
        // Merge phase
        bool my_active = false;
        if (gid < N) {
            h[gid] += temp_h[gid];
            my_active = (h[gid] > 1);
        }
        
        // Activity detection
        unsigned int lane = threadIdx.x & 31;
        uint32_t warp_mask = __ballot_sync(0xFFFFFFFFu, my_active);
        
        __shared__ bool block_active;
        if (threadIdx.x == 0) {
            block_active = false;
        }
        __syncthreads();
        
        if (lane == 0 && warp_mask != 0) {
            block_active = true;
        }
        __syncthreads();
        
        if (threadIdx.x == 0) {
            active_mask[blockIdx.x] = block_active ? 1u : 0u;
        }
    }
    
    if (gid == 0) {
        *steps_taken = max_steps;
    }
}

// Other kernels remain the same
__global__ void inicializacion_kernel(MannaItemType *h) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        h[idx] = static_cast<MannaItemType>((idx + 1) * DENSITY)
               - static_cast<MannaItemType>(idx * DENSITY);
    }
}

__global__ void desestabilizacion_kernel(
    MannaItemType *h, MannaItemType *temp_h, uint32_t *rng)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= N) return;
    if (h[idx] == 1) {
        bool dir = xorshift32(rng[idx]) & 1;
        int j = (idx + 2*dir - 1) & (N-1);
        atomicAdd(&temp_h[j], 1u);
        h[idx] = 0;
    }
}

__global__ void setup_rng(uint32_t *rng_states, uint32_t seed) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        rng_states[idx] = seed ^ (idx * 0x9E3779B1u);
    }
}

class MannaCUDA {
private:
    MannaItemType *h, *temp_h;
    uint32_t      *rng_state;
    uint32_t      *active_mask;
    int           *global_active; // GPU-based activity flag
    int           *steps_counter; // GPU-based step counter
    int            block_size, grid_size;

public:
    MannaCUDA(){
        block_size = BLOCK_SIZE < N ? BLOCK_SIZE : N;
        grid_size  = (N + block_size - 1)/ block_size;

        hipMalloc(&h,            N * sizeof(*h));
        hipMalloc(&temp_h,       N * sizeof(*temp_h));
        hipMalloc(&rng_state,    N * sizeof(*rng_state));
        hipMalloc(&active_mask,  grid_size * sizeof(*active_mask));
        hipMalloc(&global_active, sizeof(int));
        hipMalloc(&steps_counter, sizeof(int));

        setup_rng<<<grid_size,block_size>>>(rng_state, SEED);
        hipDeviceSynchronize();
    }
    
    ~MannaCUDA(){
        hipFree(h);
        hipFree(temp_h);
        hipFree(rng_state);
        hipFree(active_mask);
        hipFree(global_active);
        hipFree(steps_counter);
    }

    void inicializacion(){
        inicializacion_kernel<<<grid_size, block_size>>>(h);
        hipDeviceSynchronize();
    }

    void desestabilizacion_inicial(){
        hipMemsetAsync(temp_h, 0, N * sizeof(*temp_h));
        desestabilizacion_kernel<<<grid_size, block_size>>>(h, temp_h, rng_state);
        hipDeviceSynchronize();
    }

    // Optimized version that avoids host-device transfers
    bool descargar_opt() {
        // Clear arrays (do this less frequently or batch)
        hipMemsetAsync(temp_h, 0, N * sizeof(*temp_h));
        hipMemsetAsync(active_mask, 0, grid_size * sizeof(*active_mask));
        hipMemsetAsync(global_active, 0, sizeof(int));

        // Launch kernels
        destab_kernel_opt<<<grid_size, block_size>>>(h, temp_h, rng_state);
        merge_kernel_opt<<<grid_size, block_size>>>(h, temp_h, active_mask);

        // Check activity on GPU
        int check_blocks = (grid_size + 255) / 256;
        check_activity_kernel<<<check_blocks, 256>>>(active_mask, grid_size, global_active);

        // Only one small memory transfer
        int host_active;
        hipMemcpy(&host_active, global_active, sizeof(int), hipMemcpyDeviceToHost);
        
        return host_active != 0;
    }

    // Original version for comparison
    bool descargar() {
        hipMemsetAsync(temp_h, 0, N * sizeof(*temp_h));
        hipMemsetAsync(active_mask, 0, grid_size * sizeof(*active_mask));

        destab_kernel_opt<<<grid_size, block_size>>>(h, temp_h, rng_state);
        merge_kernel_opt<<<grid_size, block_size>>>(h, temp_h, active_mask);

        std::vector<uint32_t> host_mask(grid_size);
        hipMemcpy(host_mask.data(), active_mask, 
                   grid_size * sizeof(uint32_t), hipMemcpyDeviceToHost);
        
        for (auto &w : host_mask) {
            if (w) return true;
        }
        return false;
    }

    // Batch processing version
    int run_batched_steps(int max_batch_size) {
        hipMemset(steps_counter, 0, sizeof(int));
        
        // This is a simplified version - full implementation would need cooperative groups
        // for proper multi-block synchronization
        multi_step_kernel<<<grid_size, block_size>>>(
            h, temp_h, rng_state, active_mask, max_batch_size, steps_counter);
        
        int steps_taken;
        hipMemcpy(&steps_taken, steps_counter, sizeof(int), hipMemcpyDeviceToHost);
        return steps_taken;
    }

    void swap_arrays() {
        MannaItemType *temp = h;
        h = temp_h;
        temp_h = temp;
    }

    void print_array_h() {
        std::vector<MannaItemType> host_h(N);
        hipMemcpy(host_h.data(), h, N * sizeof(MannaItemType), hipMemcpyDeviceToHost);
        for (size_t i = 0; i < N; ++i) {
            std::cout << host_h[i] << " ";
        }
        std::cout << "\n";
    }
};

int main() {
    auto start = std::chrono::high_resolution_clock::now();
    MannaCUDA manna;

    manna.inicializacion();
    manna.desestabilizacion_inicial();
    manna.swap_arrays();

    uint32_t t = 0;
    bool active;
    
    do {
        active = manna.descargar_opt(); // Use optimized version
        ++t;
    } while (active && t < NSTEPS);

    uint32_t processed = t * N * DENSITY;

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    std::cout << "=== RESULTADOS FINALES ===\n";
    std::cout << "Steps taken: " << t << "\n";
    std::cout << "Tiempo (s): " << static_cast<double>(duration.count())/1e6 << "\n";
    std::cout << "Granos procesados: " << processed << "\n";
    std::cout << "Granos/us: " << static_cast<double>(processed)/duration.count() << "\n";
    
    return 0;
}