#include "hip/hip_runtime.h"
#include "params.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>

#if N < 65536u
typedef uint16_t MannaSizeType;
#else
typedef uint32_t MannaSizeType;
#endif

typedef uint32_t MannaItemType;

__device__ inline uint32_t xorshift32(uint32_t &state) {
    state ^= state << 13;
    state ^= state >> 17;
    state ^= state << 5;
    return state;
}

__global__ void descargar_kernel(
    MannaItemType *h, MannaItemType *temp_h,
    uint32_t *rng_states, unsigned int *active, uint32_t *processed
) {
    // TODO
}


// Kernel para inicializar RNG a partir de seed
__global__ void init_rng_kernel(uint32_t *rng_states, uint32_t seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        // Mezcla la seed con el índice para diversificar
        uint32_t st = seed ^ (idx * 0x9E3779B1u);
        // Perturba un poco
        for (int i = 0; i < 4; ++i) st = xorshift32(st);
        rng_states[idx] = st;
    }
}

__global__ void inicializacion_kernel(MannaItemType *h) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        h[idx] = static_cast<MannaItemType>((idx + 1) * DENSITY) - 
                 static_cast<MannaItemType>(idx * DENSITY);
    }
}

__global__ void desestabilizacion_kernel(
    MannaItemType *h, MannaItemType *temp_h, uint32_t *rng)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx >= N) return;
  if (h[idx] == 1) {
    bool dir = xorshift32(rng[idx]) & 1;
    int j = (idx + 2*dir - 1) & (N-1);
    atomicAdd(&temp_h[j], 1u);
  }
}
__global__ void setup_rng(uint32_t *rng_states, uint32_t seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        uint32_t st = seed ^ (idx * 0x9E3779B1u);
        rng_states[idx] = st;
    }
}


class MannaCUDA {
private:
    MannaItemType *h, *temp_h;
    uint32_t      *rng_state;
    uint32_t      *processed;
    unsigned int  *activity;
    int block_size;
    int grid_size;

public:
    MannaCUDA(){
        block_size = BLOCK_SIZE < N ? BLOCK_SIZE : N;
        grid_size = N / block_size;
        hipMalloc(&h,         N * sizeof(MannaItemType));
        hipMalloc(&temp_h,    N * sizeof(MannaItemType));
        hipMalloc(&rng_state, N * sizeof(uint32_t));
        hipMalloc(&processed, sizeof(uint32_t));
        hipMalloc(&activity,  sizeof(unsigned int));
        setup_rng<<<grid_size, block_size>>>(rng_state, SEED);
    }

    ~MannaCUDA() {
        hipFree(h);
        hipFree(temp_h);
        hipFree(rng_state);
        hipFree(processed);
        hipFree(activity);
    }
    
    void inicializacion() {
        inicializacion_kernel<<<grid_size, block_size>>>(h);
    }
    
    void desestabilizacion_inicial() {
        desestabilizacion_kernel<<<grid_size, block_size>>>(
            h, temp_h, rng_state);
    }

    // swap d_h and d_temp_h
    void swap_arrays() {
        MannaItemType *temp = h;
        h = temp_h;
        temp_h = temp;
    }
    
    bool descargar() {
        // todo
        return true;
    }

    void print_array() {
        std::vector<MannaItemType> h_host(N);
        hipMemcpy(h_host.data(), h, N * sizeof(MannaItemType), hipMemcpyDeviceToHost);
        
        std::cout << "h: ";
        for (int i = 0; i < N; ++i) {
            std::cout << h_host[i] << " ";
        }
        std::cout << std::endl;
    }
};

int main() {
    auto start = std::chrono::high_resolution_clock::now();
    MannaCUDA manna;
    manna.inicializacion();
    manna.desestabilizacion_inicial();
    manna.swap_arrays();

    // auto end_init = std::chrono::high_resolution_clock::now();
    // auto duration_init = std::chrono::duration_cast<std::chrono::microseconds>(end_init - start);
    // std::cout << "Tiempo de inicialización (s): " << static_cast<double>(duration_init.count()) / 1e6 << std::endl;
    uint32_t t = 0;
    uint32_t processed = 0;
    bool active;

    do {
        active = manna.descargar();
        //manna.print_array();
        t++;
    } while (active && t < NSTEPS);
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    
    std::cout << "=== RESULTADOS FINALES ===" << std::endl;
    std::cout << "Steps taken: " << t << std::endl;
    std::cout << "Tiempo de procesamiento (s): " << static_cast<double>(duration.count()) / 1e6 << std::endl;
    std::cout << "Granos procesados: " << processed << std::endl;
    std::cout << "Granos/us: " << static_cast<double>(processed) / duration.count() << std::endl;    
    return 0;
}