#include "hip/hip_runtime.h"
#include "params.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>

#if N < 65536u
typedef uint16_t MannaSizeType;
#else
typedef uint32_t MannaSizeType;
#endif

// Usamos uint32_t para contadores y atomics
typedef uint32_t MannaItemType;

// XORSHIFT32 para RNG rápido en registros
__device__ inline uint32_t xorshift32(uint32_t &state) {
    state ^= state << 13;
    state ^= state >> 17;
    state ^= state << 5;
    return state;
}

__global__ void descargar_kernel(
    MannaItemType *h, MannaItemType *temp_h,
    uint32_t *rng_states, unsigned int *active, uint32_t *processed
) {
    // TODO
}


// Kernel para inicializar RNG a partir de seed
__global__ void init_rng_kernel(uint32_t *rng_states, uint32_t seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        // Mezcla la seed con el índice para diversificar
        uint32_t st = seed ^ (idx * 0x9E3779B1u);
        // Perturba un poco
        for (int i = 0; i < 4; ++i) st = xorshift32(st);
        rng_states[idx] = st;
    }
}

__global__ void inicializacion_kernel(MannaItemType *h) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        h[idx] = static_cast<MannaItemType>((idx + 1) * DENSITY) - 
                 static_cast<MannaItemType>(idx * DENSITY);
    }
}

__global__ void desestabilizacion_kernel(
    MannaItemType *h, MannaItemType *temp_h, uint32_t *rng)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx >= N) return;
  if (h[idx] == 1) {
    bool dir = xorshift32(rng[idx]) & 1;
    int j = (idx + 2*dir - 1) & (N-1);
    atomicAdd(&temp_h[j], 1u);
  }
}
__global__ void setup_rng(uint32_t *rng_states, uint32_t seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        uint32_t st = seed ^ (idx * 0x9E3779B1u);
        rng_states[idx] = st;
    }
}


class MannaCUDA {
private:
    MannaItemType *d_h, *d_temp_h;
    uint32_t      *d_rng_state;
    uint32_t      *d_processed;
    unsigned int  *d_activity;
    int block_size;
    int grid_size;

public:
    MannaCUDA(){
        block_size = BLOCK_SIZE < N ? BLOCK_SIZE : N;
        grid_size = N / block_size;
        hipMalloc(&d_h,         N * sizeof(MannaItemType));
        hipMalloc(&d_temp_h,    N * sizeof(MannaItemType));
        hipMalloc(&d_rng_state, N * sizeof(uint32_t));
        hipMalloc(&d_processed, sizeof(uint32_t));
        hipMalloc(&d_activity,  sizeof(unsigned int));
        setup_rng<<<grid_size, block_size>>>(d_rng_state, SEED);
    }

    ~MannaCUDA() {
        hipFree(d_h);
        hipFree(d_temp_h);
        hipFree(d_rng_state);
        hipFree(d_processed);
        hipFree(d_activity);
    }
    
    void inicializacion() {
        inicializacion_kernel<<<grid_size, block_size>>>(d_h);
    }
    
    void desestabilizacion_inicial() {
        desestabilizacion_kernel<<<grid_size, block_size>>>(
            d_h, d_temp_h, d_rng_state);
    }

    // swap d_h and d_temp_h
    void swap_arrays() {
        MannaItemType *temp = d_h;
        d_h = d_temp_h;
        d_temp_h = temp;
    }
    
    bool descargar() {
        // todo
    }

    void print_array() {
        std::vector<MannaItemType> h_host(N);
        hipMemcpy(h_host.data(), d_h, N * sizeof(MannaItemType), hipMemcpyDeviceToHost);
        
        std::cout << "h: ";
        for (int i = 0; i < N; ++i) {
            std::cout << h_host[i] << " ";
        }
        std::cout << std::endl;
    }
};

int main() {
    auto start = std::chrono::high_resolution_clock::now();
    
    MannaCUDA manna;
    manna.inicializacion();
    manna.desestabilizacion_inicial();
    manna.swap_arrays();
    manna.print_array();
    uint32_t t = 0;
    uint32_t processed = 0;
    bool active;

    do {
        active = manna.descargar();
        //manna.print_array();
        t++;
    } while (active && t < NSTEPS);
    
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    
    std::cout << "=== RESULTADOS FINALES ===" << std::endl;
    std::cout << "Steps taken: " << t << std::endl;
    std::cout << "Tiempo de procesamiento (s): " << static_cast<double>(duration.count()) / 1e6 << std::endl;
    std::cout << "Granos procesados: " << processed << std::endl;
    std::cout << "Granos/us: " << static_cast<double>(processed) / duration.count() << std::endl;    
    return 0;
}