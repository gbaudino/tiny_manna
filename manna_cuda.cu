#include "hip/hip_runtime.h"
#include "params.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>

#if N < 65536u
typedef uint16_t MannaSizeType;
#else
typedef uint32_t MannaSizeType;
#endif

typedef uint32_t MannaItemType;

__device__ inline uint32_t xorshift32(uint32_t &state) {
    state ^= state << 13;
    state ^= state >> 17;
    state ^= state << 5;
    return state;
}

// Combined kernel that does destabilization + merge + activity check in one pass
// This maintains the EXACT same logic as the original three separate kernels
__global__ void combined_step_kernel(
    MannaItemType *h,           // Main array
    MannaItemType *temp_h,      // Temporary array
    uint32_t      *rng_state,   // RNG states
    uint32_t      *active_mask, // Block activity mask
    int           *global_active // Global activity flag
) {
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int lane = threadIdx.x & 31;
    unsigned int warp_id = threadIdx.x >> 5;
    
    // Shared memory for warp results
    __shared__ uint32_t warp_results[32];
    
    // Phase 1: Clear temp array (same as original memset)
    if (gid < N) {
        temp_h[gid] = 0;
    }
    
    // Ensure temp array is cleared before destabilization
    __syncthreads();
    
    // Phase 2: Destabilization (EXACT same logic as destab_kernel_opt)
    if (gid < N) {
        MannaItemType c = h[gid];
        
        // Early exit for inactive cells (same as original)
        if (c <= 1) {
            // Do nothing, just like original
        } else if (c >= 32) {
            // Handle overflow case (same as original)
            h[gid] = 0;
        } else {
            // Normal destabilization (same as original)
            h[gid] = 0;
            uint32_t r = xorshift32(rng_state[gid]);
            uint32_t mask = ((1u << c) - 1u) & r;
            uint32_t to_right = __popc(mask);
            uint32_t to_left = c - to_right;
            
            unsigned right = (gid + 1) & (N - 1);
            unsigned left = (gid + N - 1) & (N - 1);
            
            atomicAdd(&temp_h[right], to_right);
            atomicAdd(&temp_h[left], to_left);
        }
    }
    
    // Synchronize to ensure all destabilization is complete
    __syncthreads();
    
    // Phase 3: Merge (EXACT same logic as merge_kernel_opt)
    bool my_active = false;
    if (gid < N) {
        MannaItemType new_val = h[gid] + temp_h[gid];
        h[gid] = new_val;
        my_active = (new_val > 1);  // Same activity condition as original
        temp_h[gid] = 0; // Clear for next iteration (same as original)
    }
    
    // Phase 4: Activity detection (EXACT same logic as merge_kernel_opt)
    uint32_t warp_mask = __ballot_sync(0xFFFFFFFFu, my_active);
    
    // Store warp results in shared memory
    if (lane == 0) {
        warp_results[warp_id] = warp_mask;
    }
    __syncthreads();
    
    // Thread 0 consolidates results (same as original)
    if (threadIdx.x == 0) {
        bool block_active = false;
        int num_warps = (blockDim.x + 31) / 32;
        
        for (int i = 0; i < num_warps; i++) {
            if (warp_results[i] != 0) {
                block_active = true;
                break;
            }
        }
        
        // Update block activity mask (same as original)
        active_mask[blockIdx.x] = block_active ? 1u : 0u;
        
        // Atomically update global activity if this block is active
        if (block_active) {
            atomicOr(global_active, 1);
        }
    }
}

// Modified merge kernel that also updates global activity atomically
__global__ void merge_kernel_with_global_activity(
    MannaItemType *h,
    MannaItemType *temp_h,
    uint32_t      *active_mask,
    int           *global_active)
{
    unsigned int gid  = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int lane = threadIdx.x & 31;

    MannaItemType new_val = 0;
    bool my_active = false;
    
    // Merge step with clearing (EXACT same as original)
    if (gid < N) {
        new_val = h[gid] + temp_h[gid];
        h[gid] = new_val;
        my_active = (new_val > 1);
        temp_h[gid] = 0; // Clear for next iteration - eliminates memset!
    }

    // Fast warp-wide ballot for activity detection (same as original)
    uint32_t warp_mask = __ballot_sync(0xFFFFFFFFu, my_active);

    // Only lane 0 of warps with activity write to shared memory (same as original)
    __shared__ uint32_t warp_results[32]; // Max 32 warps per block
    unsigned int warp_id = threadIdx.x >> 5;
    
    if (lane == 0) {
        warp_results[warp_id] = warp_mask;
    }
    __syncthreads();

    // Thread 0 consolidates warp results (same as original)
    if (threadIdx.x == 0) {
        bool block_active = false;
        int num_warps = (blockDim.x + 31) / 32;
        for (int i = 0; i < num_warps; i++) {
            if (warp_results[i] != 0) {
                block_active = true;
                break;
            }
        }
        active_mask[blockIdx.x] = block_active ? 1u : 0u;
        
        // NEW: Also update global activity atomically
        if (block_active) {
            atomicOr(global_active, 1);
        }
    }
}

// Keep your original optimized destabilization kernel
__global__ void destab_kernel_opt(
    MannaItemType *h,
    MannaItemType *temp_h,
    uint32_t      *rng)
{
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= N) return;

    MannaItemType c = h[gid];
    
    // Early exit for inactive cells
    if (c <= 1) return;
    
    // Handle overflow case
    if (c >= 32) {
        h[gid] = 0;
        return;
    }

    h[gid] = 0;
    uint32_t r = xorshift32(rng[gid]);
    uint32_t mask = ((1u << c) - 1u) & r;
    uint32_t to_right = __popc(mask);
    uint32_t to_left  = c - to_right;

    unsigned right = (gid + 1) & (N - 1);
    unsigned left  = (gid + N - 1) & (N - 1);

    atomicAdd(&temp_h[right], to_right);
    atomicAdd(&temp_h[left],  to_left);
}
__global__ void multi_step_combined_kernel(
    MannaItemType *h,
    MannaItemType *temp_h,
    uint32_t      *rng_state,
    uint32_t      *active_mask,
    int           *global_active,
    int           *step_counter,
    int           max_steps
) {
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int lane = threadIdx.x & 31;
    unsigned int warp_id = threadIdx.x >> 5;
    
    __shared__ uint32_t warp_results[32];
    __shared__ bool continue_flag;
    
    for (int step = 0; step < max_steps; step++) {
        // Reset global activity flag
        if (gid == 0) {
            *global_active = 0;
            continue_flag = true;
        }
        __syncthreads();
        
        // Check if we should continue (after first step)
        if (step > 0 && gid == 0) {
            // Simple check - if no activity detected in previous step, stop
            if (*global_active == 0) {
                continue_flag = false;
                *step_counter = step;
            }
        }
        __syncthreads();
        
        if (!continue_flag) break;
        
        // Clear temp array and destabilize
        if (gid < N) {
            temp_h[gid] = 0;
            
            MannaItemType c = h[gid];
            if (c > 1) {
                if (c >= 32) {
                    h[gid] = 0;
                } else {
                    h[gid] = 0;
                    uint32_t r = xorshift32(rng_state[gid]);
                    uint32_t mask = ((1u << c) - 1u) & r;
                    uint32_t to_right = __popc(mask);
                    uint32_t to_left = c - to_right;
                    
                    unsigned right = (gid + 1) & (N - 1);
                    unsigned left = (gid + N - 1) & (N - 1);
                    
                    atomicAdd(&temp_h[right], to_right);
                    atomicAdd(&temp_h[left], to_left);
                }
            }
        }
        __syncthreads();
        
        // Merge and check activity
        bool my_active = false;
        if (gid < N) {
            MannaItemType new_val = h[gid] + temp_h[gid];
            h[gid] = new_val;
            my_active = (new_val > 1);
        }
        
        // Activity detection
        uint32_t warp_mask = __ballot_sync(0xFFFFFFFFu, my_active);
        
        if (lane == 0) {
            warp_results[warp_id] = warp_mask;
        }
        __syncthreads();
        
        if (threadIdx.x == 0) {
            bool block_active = false;
            int num_warps = (blockDim.x + 31) / 32;
            
            for (int i = 0; i < num_warps; i++) {
                if (warp_results[i] != 0) {
                    block_active = true;
                    break;
                }
            }
            
            active_mask[blockIdx.x] = block_active ? 1u : 0u;
            
            if (block_active) {
                atomicOr(global_active, 1);
            }
        }
        __syncthreads();
    }
    
    // Update final step counter
    if (gid == 0 && continue_flag) {
        *step_counter = max_steps;
    }
}

// Remaining original kernels
__global__ void inicializacion_kernel(MannaItemType *h) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        h[idx] = static_cast<MannaItemType>((idx + 1) * DENSITY)
               - static_cast<MannaItemType>(idx * DENSITY);
    }
}

__global__ void desestabilizacion_kernel(
    MannaItemType *h, MannaItemType *temp_h, uint32_t *rng)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= N) return;
    if (h[idx] == 1) {
        bool dir = xorshift32(rng[idx]) & 1;
        int j = (idx + 2*dir - 1) & (N-1);
        atomicAdd(&temp_h[j], 1u);
        h[idx] = 0;
    }
}

__global__ void setup_rng(uint32_t *rng_states, uint32_t seed) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        rng_states[idx] = seed ^ (idx * 0x9E3779B1u);
    }
}

class MannaCUDA {
private:
    MannaItemType *h, *temp_h;
    uint32_t      *rng_state;
    uint32_t      *active_mask;
    int           *global_active;
    int           *step_counter;
    int            block_size, grid_size;

public:
    MannaCUDA(){
        block_size = BLOCK_SIZE < N ? BLOCK_SIZE : N;
        grid_size  = (N + block_size - 1)/ block_size;

        hipMalloc(&h,            N * sizeof(*h));
        hipMalloc(&temp_h,       N * sizeof(*temp_h));
        hipMalloc(&rng_state,    N * sizeof(*rng_state));
        hipMalloc(&active_mask,  grid_size * sizeof(*active_mask));
        hipMalloc(&global_active, sizeof(int));
        hipMalloc(&step_counter, sizeof(int));

        setup_rng<<<grid_size,block_size>>>(rng_state, SEED);
        hipDeviceSynchronize();
    }
    
    ~MannaCUDA(){
        hipFree(h);
        hipFree(temp_h);
        hipFree(rng_state);
        hipFree(active_mask);
        hipFree(global_active);
        hipFree(step_counter);
    }

    void inicializacion(){
        inicializacion_kernel<<<grid_size, block_size>>>(h);
        hipDeviceSynchronize();
    }

    void desestabilizacion_inicial(){
        hipMemsetAsync(temp_h, 0, N * sizeof(*temp_h));
        desestabilizacion_kernel<<<grid_size, block_size>>>(h, temp_h, rng_state);
        hipDeviceSynchronize();
    }

    // Ultra-optimized single kernel approach - maintains exact original logic
    bool descargar_combined() {
        // Reset global activity flag (same as original memset)
        hipMemsetAsync(global_active, 0, sizeof(int));
        
        // Single kernel that does everything with EXACT same logic as original 3 kernels
        combined_step_kernel<<<grid_size, block_size>>>(
            h, temp_h, rng_state, active_mask, global_active);
        
        // Single memory transfer (replaces the original check_activity_kernel_fast + memcpy)
        int host_active;
        hipMemcpy(&host_active, global_active, sizeof(int), hipMemcpyDeviceToHost);
        
        return host_active != 0;
    }
    
    // Conservative optimization: Keep original kernels but optimize activity check
    bool descargar_optimized_activity() {
        // Clear arrays (same as original)
        hipMemsetAsync(temp_h, 0, N * sizeof(*temp_h));
        hipMemsetAsync(global_active, 0, sizeof(int));
        
        // Use your original optimized kernels
        destab_kernel_opt<<<grid_size, block_size>>>(h, temp_h, rng_state);
        
        // Modified merge kernel that also updates global activity
        merge_kernel_with_global_activity<<<grid_size, block_size>>>(
            h, temp_h, active_mask, global_active);
        
        // Single memory transfer instead of check_activity_kernel_fast + memcpy
        int host_active;
        hipMemcpy(&host_active, global_active, sizeof(int), hipMemcpyDeviceToHost);
        
        return host_active != 0;
    }
    
    // Batch processing with reduced transfers
    int run_batch_optimized(int batch_size = 100) {
        // Reset counters
        hipMemset(global_active, 0, sizeof(int));
        hipMemset(step_counter, 0, sizeof(int));
        
        // Process multiple steps in single kernel launch
        multi_step_combined_kernel<<<grid_size, block_size>>>(
            h, temp_h, rng_state, active_mask, global_active, step_counter, batch_size);
        
        // Single memory transfer to get step count
        int steps_taken;
        hipMemcpy(&steps_taken, step_counter, sizeof(int), hipMemcpyDeviceToHost);
        
        return steps_taken;
    }
    
    // Adaptive batch processing
    int run_adaptive_batch() {
        int total_steps = 0;
        int batch_size = 1000;  // Start with larger batches
        
        while (total_steps < NSTEPS) {
            int remaining = NSTEPS - total_steps;
            int current_batch = (remaining < batch_size) ? remaining : batch_size;
            
            int steps_taken = run_batch_optimized(current_batch);
            total_steps += steps_taken;
            
            // If we took fewer steps than the batch size, we're done
            if (steps_taken < current_batch) {
                break;
            }
            
            // Adaptive batch sizing - increase batch size for longer runs
            if (steps_taken == current_batch && batch_size < 10000) {
                batch_size *= 2;
            }
        }
        
        return total_steps;
    }

    // Original methods for comparison
    bool descargar_ultra_opt() {
        hipMemsetAsync(active_mask, 0, grid_size * sizeof(*active_mask));
        hipMemsetAsync(global_active, 0, sizeof(int));

        // These would be your original optimized kernels
        // destab_kernel_opt<<<grid_size, block_size>>>(h, temp_h, rng_state);
        // merge_kernel_opt<<<grid_size, block_size>>>(h, temp_h, active_mask);
        // check_activity_kernel_fast<<<1, 32>>>(active_mask, grid_size, global_active);

        int host_active;
        hipMemcpy(&host_active, global_active, sizeof(int), hipMemcpyDeviceToHost);
        
        return host_active != 0;
    }

    void swap_arrays() {
        MannaItemType *temp = h;
        h = temp_h;
        temp_h = temp;
    }

    void print_array_h() {
        std::vector<MannaItemType> host_h(N);
        hipMemcpy(host_h.data(), h, N * sizeof(MannaItemType), hipMemcpyDeviceToHost);
        for (size_t i = 0; i < N; ++i) {
            std::cout << host_h[i] << " ";
        }
        std::cout << "\n";
    }
};

int main() {
    auto start = std::chrono::high_resolution_clock::now();
    MannaCUDA manna;

    manna.inicializacion();
    manna.desestabilizacion_inicial();
    manna.swap_arrays();

    // Option 1: Conservative optimization (2 kernels instead of 3)
    uint32_t t = 0;
    bool active;    
    do {
        active = manna.descargar_optimized_activity(); // 2 kernels per iteration
        ++t;
    } while (active && t < NSTEPS);
    
    /*
    // Option 2: Single kernel optimization (most aggressive but safe)
    uint32_t t = 0;
    bool active;    
    do {
        active = manna.descargar_combined(); // 1 kernel per iteration
        ++t;
    } while (active && t < NSTEPS);
    */
    
    /*
    // Option 3: Batch processing (experimental - use only if above work correctly)
    uint32_t t = manna.run_adaptive_batch();
    */

    uint32_t processed = t * N * DENSITY;

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    std::cout << "=== RESULTADOS FINALES ===\n";
    std::cout << "Steps taken: " << t << "\n";
    std::cout << "Tiempo (s): " << static_cast<double>(duration.count())/1e6 << "\n";
    std::cout << "Granos procesados: " << processed << "\n";
    std::cout << "Granos/us: " << static_cast<double>(processed)/duration.count() << "\n";
    
    return 0;
}